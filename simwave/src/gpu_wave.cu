#include "hip/hip_runtime.h"
#include "mpi.h"
#include "wave.h"
#include "memory.h"
#include "math.h"
#include "timer.h"
#include "velocity.h"
#include "source.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define U0(z,y,x)   (w->u0[(x+w->sx) + (2*w->sx + w->dimx) * \
                           ((2*w->sy + w->dimy) * (z+w->sz) + (y+w->sy))])
#define U1(z,y,x)   (w->u1[(x+w->sx) + (2*w->sx + w->dimx) * \
                           ((2*w->sy + w->dimy) * (z+w->sz) + (y+w->sy))])


#define ROC2(z,y,x) (w->roc2[x + (w->dimx * (w->dimy*(z) + y))])
#define PHI(z,y,x)  ( w->phi[x + (w->dimx * (w->dimy*(z) + y))])
#define ETA(z,y,x)  (w->eta[(2+w->dimx)*((2+w->dimy)*(z+1) + (y+1)) + (x+1)])


#define PWAVE_COMPUTE_LAPLACIAN() \
	laplacian = w->coef0 * U0(z, y, x)                          \
        + w->coefx[1]*( U0(z,   y,   x+1) + U0(z,   y,   x-1))  \
        + w->coefy[1]*( U0(z,   y+1, x  ) + U0(z,   y-1, x  ))  \
        + w->coefz[1]*( U0(z+1, y,   x  ) + U0(z-1, y,   x  ))  \
        + w->coefx[2]*( U0(z,   y,   x+2) + U0(z,   y,   x-2))  \
        + w->coefy[2]*( U0(z,   y+2, x  ) + U0(z,   y-2, x  ))  \
        + w->coefz[2]*( U0(z+2, y,   x  ) + U0(z-2, y,   x  ))  \
        + w->coefx[3]*( U0(z,   y,   x+3) + U0(z,   y,   x-3))  \
        + w->coefy[3]*( U0(z,   y+3, x  ) + U0(z,   y-3, x  ))  \
        + w->coefz[3]*( U0(z+3, y,   x  ) + U0(z-3, y,   x  ))  \
        + w->coefx[4]*( U0(z,   y,   x+4) + U0(z,   y,   x-4))  \
        + w->coefy[4]*( U0(z,   y+4, x  ) + U0(z,   y-4, x  ))  \
        + w->coefz[4]*( U0(z+4, y,   x  ) + U0(z-4, y,   x  ));

#define PWAVE_UPDATE_INNER_FIELDS()                                  \
    U1(z,y,x) = 2.*U0(z,y,x) - U1(z,y,x) + ROC2(z,y,x) * laplacian;

#define PWAVE_UPDATE_PML_FIELDS()                                                        \
    U1(z,y,x) = ((2.-ETA(z,y,x)*ETA(z,y,x) + 2.*ETA(z,y,x))*U0(z,y,x)                    \
                 - U1(z,y,x) + ROC2(z,y,x)*(laplacian + PHI(z,y,x)))/(1.+2.*ETA(z,y,x)); \
    PHI(z,y,x)= (PHI(z,y,x)-                                                             \
                 (( ETA(z,   y,   x+1) - ETA(z,   y,   x-1))                             \
                  *( U0(z,   y,   x+1) -  U0(z,   y,   x-1))*w->hdx2                     \
                  +(ETA(z,   y+1, x  ) - ETA(z,   y-1, x  ))                             \
                  *( U0(z,   y+1, x  ) -  U0(z,   y-1, x  ))*w->hdy2                     \
                  +(ETA(z+1, y,   x  ) - ETA(z-1, y,   x  ))                             \
                  *( U0(z+1, y,   x  ) -  U0(z-1, y,   x  ))*w->hdz2))/(1.+ETA(z,y,x));

/*
__global__ void gpu_pwave_swap_pointers(pwave_t* w) {
    DATATYPE* tmp = w->u0;
    w->u0 = w->u1;
    w->u1 = tmp;
}
*/

INLINE void pwave_snapshot(pwave_t* w) {
    unsigned int x, y;
    size_t size = sizeof(DATATYPE);
    for (x = 0; x < w->dimx; x++)
        for (y = 0; y < w->dimy; y++)
            fwrite(&U0(w->sourcez, y, x), size, 1, w->snapshot_file);
}
/*
int checkcpy(pwave_t *w,pwave_t *gw){
	if (!(gw->dimx == w->dimx &&
	gw->dimy == w->dimy &&
	gw->dimz == w->dimz &&
	gw->dx == w->dx &&
	gw->dy == w->dy &&
	gw->dz == w->dz &&
	gw->sx == w->sx &&
	gw->sy == w->sy &&
	gw->sz == w->sz &&
	gw->pmlx == w->pmlx &&
	gw->pmly == w->pmly &&
	gw->pmlz == w->pmlz &&
	gw->sourcex == w->sourcex &&
	gw->sourcey == w->sourcey &&
	gw->sourcez == w->sourcez &&
	gw->time_steps == w->time_steps &&

	gw->coef0 == w->coef0 &&
	gw->hdx2 == w->hdx2 &&
	gw->hdy2 == w->hdy2 &&
	gw->hdz2 == w->hdz2 &&
	gw->dt == w->dt &&
	gw->lambda == w->lambda)) return -1;
	size_t i;
	for(i=0;i<w->size;i++){
		if(w->u0[i]!=gw->u0[i] || w->u1[i]!=gw->u1[i] ) return -1;
	}

	for(i=0;i< w->dimx * w->dimy * w->dimz;i++){
		if(w->phi[i]!=gw->phi[i] || w->roc2[i]!=gw->roc2[i] ) return -1;
	}
	return 1;
}
*/

pwave_t* myCudaMemcpy(pwave_t *w, pwave_t **hw){

	pwave_t * gpuw;
	pwave_t * hd = (pwave_t*)malloc( sizeof(pwave_t));

	hipMalloc((void**)&gpuw,sizeof(pwave_t));

	hd->size = w->size;
	hd->dimx = w->dimx;
	hd->dimy = w->dimy;
	hd->dimz = w->dimz;
	hd->dx = w->dx;
	hd->dy = w->dy;
	hd->dz = w->dz;
	hd->sx = w->sx;
	hd->sy = w->sy;
	hd->sz = w->sz;
	hd->pmlx = w->pmlx;
	hd->pmly = w->pmly;
	hd->pmlz = w->pmlz;
	hd->sourcex = w->sourcex;
	hd->sourcey = w->sourcey;
	hd->sourcez = w->sourcez;
	hd->time_steps = w->time_steps;

	hd->coef0 = w->coef0;
	hd->hdx2 = w->hdx2;
	hd->hdy2 = w->hdy2;
	hd->hdz2 = w->hdz2;
	hd->dt = w->dt;
	hd->lambda = w->lambda;


/************************* allouer et copier les tableaux internes ***************************************/

	size_t taille = sizeof(DATATYPE)*w->size;
//	hipError_t error;

	hipMalloc((void**)&(hd->u0),taille);				hipMemcpy(hd->u0,w->u0,taille,hipMemcpyHostToDevice);
	hipMalloc((void**)&(hd->u1),taille);				hipMemcpy(hd->u1,w->u1,taille,hipMemcpyHostToDevice);

	taille = w->dimx * w->dimy * w->dimz * sizeof(DATATYPE);
	hipMalloc((void**)&(hd->roc2),taille);				hipMemcpy(hd->roc2,w->roc2,taille,hipMemcpyHostToDevice);
	hipMalloc((void**)&(hd->phi),taille);				hipMemcpy(hd->phi,w->phi,taille,hipMemcpyHostToDevice);

	taille = w->time_steps*sizeof(DATATYPE);
	hipMalloc((void**)&(hd->source),taille);			hipMemcpy(hd->source,w->source,taille,hipMemcpyHostToDevice);

	taille = (2*w->sx + 1)*sizeof(DATATYPE);
	hipMalloc((void**)&(hd->coefx),taille);			hipMemcpy(hd->coefx,w->coefx,taille,hipMemcpyHostToDevice);
	taille = (2*w->sy + 1)*sizeof(DATATYPE);
	hipMalloc((void**)&(hd->coefy),taille);			hipMemcpy(hd->coefy,w->coefy,taille,hipMemcpyHostToDevice);
	taille = (2*w->sz + 1)*sizeof(DATATYPE);
	hipMalloc((void**)&(hd->coefz),taille);			hipMemcpy(hd->coefz,w->coefz,taille,hipMemcpyHostToDevice);

	taille = (w->dimx+2)*(w->dimy+2)*(w->dimz+2)*sizeof(DATATYPE);
	hipMalloc((void**)&(hd->eta),taille);				hipMemcpy(hd->eta,w->eta,taille,hipMemcpyHostToDevice);

//	hd->snapshot_file = w->snapshot_file;

/************************* copier hd sur le gpu pour recuperer les pointers *****************************/
	hipMemcpy(gpuw,hd,sizeof(pwave_t),hipMemcpyHostToDevice);

	(*hw)= hd;
return gpuw;
}



__global__ void gpu_pwave_update_source_swap_pointers(pwave_t* w, unsigned int time_step) {

	U1(w->sourcez, w->sourcey, w->sourcex) = U1(w->sourcez, w->sourcey, w->sourcex)  + w->source[time_step];
	DATATYPE* tmp = w->u0;
	w->u0 = w->u1;
	w->u1 = tmp;
}

__global__ void gpu_pwave_update_fields_3D(pwave_t *w,int t) {


	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	int z = blockDim.z*blockIdx.z+threadIdx.z;
	DATATYPE laplacian;

	if((x < w->dimx) && (y < w->dimy) && (z < w->dimz)){
//		PWAVE_COMPUTE_LAPLACIAN();


		float l1 = w->coef0 * U0(z, y, x);
		float l2 = w->coefx[1]*( U0(z,   y,   x+1) + U0(z,   y,   x-1));
		float l3 = w->coefy[1]*( U0(z,   y+1, x  ) + U0(z,   y-1, x  ));
		float l4 = w->coefz[1]*( U0(z+1, y,   x  ) + U0(z-1, y,   x  ));
		float l5 = w->coefx[2]*( U0(z,   y,   x+2) + U0(z,   y,   x-2));
		float l6 = w->coefy[2]*( U0(z,   y+2, x  ) + U0(z,   y-2, x  ));
		float l7 = w->coefz[2]*( U0(z+2, y,   x  ) + U0(z-2, y,   x  ));
		float l8 = w->coefx[3]*( U0(z,   y,   x+3) + U0(z,   y,   x-3));
		float l9 = w->coefy[3]*( U0(z,   y+3, x  ) + U0(z,   y-3, x  ));
		float l10 = w->coefz[3]*( U0(z+3, y,   x  ) + U0(z-3, y,   x  ));
		float l11 = w->coefx[4]*( U0(z,   y,   x+4) + U0(z,   y,   x-4));
		float l12 = w->coefy[4]*( U0(z,   y+4, x  ) + U0(z,   y-4, x  ));
		float l13 = w->coefz[4]*( U0(z+4, y,   x  ) + U0(z-4, y,   x  ));

	int ind = (x+w->sx) + (2*w->sx + w->dimx) * \
                         ((2*w->sy + w->dimy) * (z+w->sz) + (y+w->sy));
		if( ind == 1 ){
//			printf("%3.45f + %3.45f = %3.45f\n",l1,l2,l1+l2);
			printf("check\n");
		}


	l1 = l1+l2+l3;
	l2 = l4+l5+l6;
	l3 = l7+l8+l9;
	l4 = l10+l11+l12+l13;
	laplacian = l1+l2+l3+l4;

		if((z >= w->pmlz) && (z < w->dimz - w->pmlz) &&
		(y >= w->pmly) && (y < w->dimy - w->pmly) &&
		(x >= w->pmlx) && (x < w->dimx - w->pmlx)) {
			PWAVE_UPDATE_INNER_FIELDS();
		} else {
                   PWAVE_UPDATE_PML_FIELDS();
		}
	}

}


__global__ void gpu_pwave_update_fields_2D(pwave_t *w,int t) {

	DATATYPE laplacian;
	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	int z;


	for (z=0;z<w->dimz;z++){
		if((x < w->dimx) && (y < w->dimy)){
//			PWAVE_COMPUTE_LAPLACIAN();
		float l1 = w->coef0 * U0(z, y, x);
		float l2 = w->coefx[1]*( U0(z,   y,   x+1) + U0(z,   y,   x-1));
		float l3 = w->coefy[1]*( U0(z,   y+1, x  ) + U0(z,   y-1, x  ));
		float l4 = w->coefz[1]*( U0(z+1, y,   x  ) + U0(z-1, y,   x  ));
		float l5 = w->coefx[2]*( U0(z,   y,   x+2) + U0(z,   y,   x-2));
		float l6 = w->coefy[2]*( U0(z,   y+2, x  ) + U0(z,   y-2, x  ));
		float l7 = w->coefz[2]*( U0(z+2, y,   x  ) + U0(z-2, y,   x  ));
		float l8 = w->coefx[3]*( U0(z,   y,   x+3) + U0(z,   y,   x-3));
		float l9 = w->coefy[3]*( U0(z,   y+3, x  ) + U0(z,   y-3, x  ));
		float l10 = w->coefz[3]*( U0(z+3, y,   x  ) + U0(z-3, y,   x  ));
		float l11 = w->coefx[4]*( U0(z,   y,   x+4) + U0(z,   y,   x-4));
		float l12 = w->coefy[4]*( U0(z,   y+4, x  ) + U0(z,   y-4, x  ));
		float l13 = w->coefz[4]*( U0(z+4, y,   x  ) + U0(z-4, y,   x  ));

		int ind = (x+w->sx) + (2*w->sx + w->dimx) * \
                         ((2*w->sy + w->dimy) * (z+w->sz) + (y+w->sy));
		if( ind == 1 ){
//			printf("%3.45f + %3.45f = %3.45f\n",l1,l2,l1+l2);
			printf("check\n");
		}
		l1 = l1+l2+l3;
		l2 = l4+l5+l6;
		l3 = l7+l8+l9;
		l4 = l10+l11+l12+l13;
		laplacian = l1+l2+l3+l4;


		if((z >= w->pmlz) && (z < w->dimz - w->pmlz) &&
			(y >= w->pmly) && (y < w->dimy - w->pmly) &&
			(x >= w->pmlx) && (x < w->dimx - w->pmlx)) {
				PWAVE_UPDATE_INNER_FIELDS();
		} else {
				PWAVE_UPDATE_PML_FIELDS();
			}
		}
	}
}


extern "C"
float pwave_multiple_update_fields(pwave_t *w,int nbThreads_x,int nbThreads_y,int nbThreads_z,bool_t snapshot_enabled,int nb_snap) {


    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


	int tx, ty, tz;
	tx =  w->dimx;
 	ty =  w->dimy;
 	tz =  w->dimz;

	int taille_grille_x = tx%nbThreads_x ? tx/nbThreads_x+1:tx/nbThreads_x;
	int taille_grille_y = ty%nbThreads_y ? ty/nbThreads_y+1:ty/nbThreads_y;
	int taille_grille_z = tz%nbThreads_z ? tz/nbThreads_z+1:tz/nbThreads_z;

	dim3 threadsParBloc3D(nbThreads_x,nbThreads_y,nbThreads_z);
	dim3 threadsParBloc2D(nbThreads_x,nbThreads_y);
	dim3 tailleGrille3D(taille_grille_x,taille_grille_y,taille_grille_z);
	dim3 tailleGrille2D(taille_grille_x,taille_grille_y);

	/* creer w sur le gpu*/
	pwave_t *hw = (pwave_t*)malloc(sizeof(pwave_t));

	pwave_t *gpuw = myCudaMemcpy(w,&hw);

	int t;

//printf("----------------------------------------- DEBUT KERNEL CUDA -------------------------------\n\n");
//	hipError_t error;
//    printf("Occupancy calculator elapsed time:  %3.3f ms \n", time);

 	for(t = 0; t < w->time_steps; ++t) {

		gpu_pwave_update_fields_3D<<<tailleGrille3D,threadsParBloc3D>>>(gpuw,t);
//		gpu_pwave_update_fields_2D<<<tailleGrille2D,threadsParBloc2D>>>(gpuw,t);
		gpu_pwave_update_source_swap_pointers<<<1,1>>>(gpuw,t);


		if((snapshot_enabled) && (t%nb_snap == 0)){
			hipMemcpy(w->u0, hw->u0, sizeof(DATATYPE)*w->size ,hipMemcpyDeviceToHost);
			pwave_snapshot(w);
		}
	}
	hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

	//  recopier pour le check
    hipMemcpy(w->u0, hw->u0, sizeof(DATATYPE) *w->size ,hipMemcpyDeviceToHost);


//printf("--------------------------------------- FIN KERNEL CUDA <<----ERREUR D ARRONDIS---->>-------------------\n");
/*
	hipFree(gpuw->u0);
	hipFree(gpuw->u1);
	hipFree(gpuw->coefx);
	hipFree(gpuw->coefy);
	hipFree(gpuw->coefz);
	hipFree(gpuw->roc2);
	hipFree(gpuw->phi);
	hipFree(gpuw->eta);
	hipFree(gpuw);
	free(hw);
*/
	return time;
}












